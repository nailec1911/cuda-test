#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <chrono>
#include <iomanip>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
 int N = 1<<29;
 float *x, *y;

 // Allocate Unified Memory – accessible from CPU or GPU
 hipMallocManaged(&x, N*sizeof(float));
 hipMallocManaged(&y, N*sizeof(float));

 // initialize x and y arrays on the host
 for (int i = 0; i < N; i++) {
   x[i] = 1.0f;
   y[i] = 2.0f;
 }

 int deviceID=0;
 hipMemPrefetchAsync((const void *)x, N*sizeof(float), deviceID) ;
 hipMemPrefetchAsync((const void *)y, N*sizeof(float), deviceID) ;

 std::chrono::time_point<std::chrono::high_resolution_clock> start_time = std::chrono::high_resolution_clock::now();

 int blockSize = 256;
 int numBlocks = (N + blockSize - 1) / blockSize;
 // Run kernel on 1M elements on the GPU
 add<<<numBlocks, blockSize>>>(N, x, y);

 // Wait for GPU to finish before accessing on host
 hipDeviceSynchronize();

 std::chrono::time_point<std::chrono::high_resolution_clock> end_time = std::chrono::high_resolution_clock::now();
 std::chrono::duration<double> elapsed = end_time - start_time;
 std::cout << " Elapsed time is : " << std::setprecision(5) << elapsed.count() << " (sec) " << std::endl;

 // Check for errors (all values should be 3.0f)
 float maxError = 0.0f;
 for (int i = 0; i < N; i++) {
   maxError = fmax(maxError, fabs(y[i]-3.0f));
 }
 std::cout << "Max error: " << maxError << std::endl;

 // Free memory
 hipFree(x);
 hipFree(y);
  return 0;
}
